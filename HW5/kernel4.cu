#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 8

__device__ int mandel(float c_re, float c_im, int count)
{
    float z_re = c_re, z_im = c_im;
    int i;

    for (i = 0; i < count; ++i) {
        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    return i;
}

__global__ void mandelKernel(int *device_out, float lowerX, float lowerY, float stepX, float stepY, int resX, int maxIterations) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

    // Calculate the index of the current thread
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    int index = thisY * resX + thisX;    

    float x = lowerX + thisX * stepX;
    float y = lowerY + thisY * stepY;

    device_out[index] = mandel(x, y, maxIterations);
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int *device_out;
    int size = resX * resY * sizeof(int);

    // Allocate memory on the host and device
    // 1600 * 1200
    hipMalloc((void **)&device_out, size);

    // Launch the kernel
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks(resX / threadsPerBlock.x, resY / threadsPerBlock.y);
    mandelKernel<<<numBlocks, threadsPerBlock>>>(device_out, lowerX, lowerY, stepX, stepY, resX, maxIterations);

    // Copy the result back to the host
    hipMemcpy(img, device_out, size, hipMemcpyDeviceToHost);

    // Free the memory
    hipFree(device_out);
}
