#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 16
#define GROUP_SIZE 5

__device__ int mandel(float c_re, float c_im, int count)
{
    float z_re = c_re, z_im = c_im;
    int i;

    for (i = 0; i < count; ++i) {
        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    return i;
}

__global__ void mandelKernel(int *device_out, size_t pitch,float lowerX, float lowerY, float stepX, float stepY, int resX, int maxIterations) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

    // Calculate the index of the current thread
    int thisX = (blockIdx.x * blockDim.x + threadIdx.x) * GROUP_SIZE;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;

    for (int i = 0; i < GROUP_SIZE; i++) {
        float x = lowerX + (thisX + i) * stepX;
        float y = lowerY + thisY * stepY;

        // Calculate the pixel's value
        int *row = (int *)((char *)device_out + thisY * pitch);
        row[thisX + i] = mandel(x, y, maxIterations);
    }
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int *host_out, *device_out; // Result on host and device
    int size = resX * resY * sizeof(int);

    // Allocate memory on host and device
    size_t pitch;
    hipHostAlloc((void **)&host_out, size, hipHostMallocDefault);
    hipMallocPitch((void **)&device_out, &pitch, resX * sizeof(int), resY);

    // CUDA function
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks(resX / (threadsPerBlock.x * GROUP_SIZE), resY / threadsPerBlock.y);
    mandelKernel<<<numBlocks, threadsPerBlock>>>(device_out, pitch, lowerX, lowerY, stepX, stepY, resX, maxIterations);
    
    // Copy the result back to the host
    hipMemcpy2D(host_out, resX * sizeof(int), device_out, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost);
    memcpy(img, host_out, size);

    // Free allocated memory
    hipHostFree(host_out);
    hipFree(device_out);
}